#include "hip/hip_runtime.h"

#include "Go100x/kernels.hpp"

//======================================================================================//
// dummy cuda kernel
//
__global__
void calculateKernel(const float* input_a, const float* input_b, float* output, int size)
{

}

//======================================================================================//
// launch the kernel
//
void gpu_calculate(int block, int ngrid, const float* input_a, const float* input_b,
                   float* output, int size)
{
    calculateKernel<<<ngrid, block>>>(input_a, input_b, output, size);
}
