#include "hip/hip_runtime.h"

#include "Go100x/kernels.hpp"

//======================================================================================//
// dummy cuda kernel
//
__global__
void calculateKernel(const float* input_a, const float* input_b, float* output, int size)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < size; i += stride)
  {
    output[i] = input_a[i] * input_b[i];
  }
}

//======================================================================================//
// launch the kernel
//
void gpu_calculate(int block, int ngrid, const float* input_a, const float* input_b,
                   float* output, int size)
{
  float* input_a_d, *input_b_d, *output_d;
  hipMalloc(&input_a_d, size*sizeof(float));
  hipMalloc(&input_b_d, size*sizeof(float));
  hipMalloc(&output_d, size*sizeof(float));
  
  hipMemcpy(input_a_d, input_a, size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(input_b_d, input_b, size*sizeof(float), hipMemcpyHostToDevice);
  
  calculateKernel<<<ngrid, block>>>(input_a, input_b, output, size);
  
  hipDeviceSynchronize();

  hipFree(&input_a_d);
  hipFree(&input_b_d);
  hipMemcpy(output, output_d, size*sizeof(float), hipMemcpyDeviceToHost);
  hipFree(&output_d);
}
